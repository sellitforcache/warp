#include <optix_world.h>
#include "datadef.h"

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(intersection_point, payload, rtPayload, );
rtDeclareVariable(uint, launch_index_in, rtLaunchIndex, );
rtDeclareVariable(unsigned,  trace_type, , );
rtDeclareVariable(unsigned,  outer_cell, , );
rtBuffer<unsigned,1>          rxn_buffer;
rtBuffer<spatial_data,1>      positions_buffer;
rtBuffer<unsigned,1>      	  matnum_buffer;
rtBuffer<unsigned,1>      	  cellnum_buffer;
rtBuffer<unsigned,1>      	  remap_buffer;

RT_PROGRAM void miss()
{
	unsigned launch_index;
	
	// remap if transport
	if(trace_type==2){
		launch_index=remap_buffer[launch_index_in];
		//rtPrintf("!!!MISS IN TRANSPORT!!! rxn = %u outer_cell = %d launch_index = %d remapped index %u trace %d (x,y,z,xhat,yhat,zhat-source)= (% 10.9E % 10.9E % 10.9E)(% 10.9E % 10.9E % 10.9E)\n", rxn_buffer[launch_index_in], outer_cell, launch_index_in, launch_index, trace_type,positions_buffer[launch_index].x,positions_buffer[launch_index].y,positions_buffer[launch_index].z,positions_buffer[launch_index].xhat,positions_buffer[launch_index].yhat,positions_buffer[launch_index].zhat);
		rtPrintf("!!!MISS(x,y,z,xhat,yhat,zhat-source) = % 10.9E % 10.9E % 10.9E % 10.9E % 10.9E % 10.9E\n", positions_buffer[launch_index].x,positions_buffer[launch_index].y,positions_buffer[launch_index].z,positions_buffer[launch_index].xhat,positions_buffer[launch_index].yhat,positions_buffer[launch_index].zhat);
	}
	else{
		launch_index = launch_index_in;  // misses in fissile query and geometry plotting are expected for out-of-bounds regions in non-rectangular geometries
	}
	
	payload.sense 				= -9;
	rxn_buffer[launch_index_in]	=  997;     //miss code, same as leak basically
	payload.surf_dist 			= -1.0;
	payload.cell 				=  3000;
	payload.mat  				=  3000;
	payload.fiss 				=  0;


}
